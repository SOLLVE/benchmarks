#include "hip/hip_runtime.h"
/*********************************************************************************
Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
All rights reserved.
  
Permission to use, copy, modify and distribute this software and its documentation for 
educational purpose is hereby granted without fee, provided that the above copyright 
notice and this permission notice appear in all copies of this software and that you do 
not sell the software.
  
THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
OTHERWISE.

The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
**********************************************************************************/
#ifndef _KERNEL_H_
#define _KERNEL_H_

__global__ void
#if defined (CUDA_HYB)
Kernel( Node* g_graph_nodes, unsigned long long* g_graph_edges, bool* g_graph_mask, bool* g_updating_graph_mask, bool *g_graph_visited, int* g_cost, unsigned long long no_of_nodes, unsigned long long* g_graph_edges_2, unsigned long long edge_dev_size) 
#else
Kernel( Node* g_graph_nodes, unsigned long long* g_graph_edges, bool* g_graph_mask, bool* g_updating_graph_mask, bool *g_graph_visited, int* g_cost, unsigned long long no_of_nodes) 
#endif
{
	unsigned long long tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if( tid<no_of_nodes && g_graph_mask[tid])
	{
		g_graph_mask[tid]=false;
		for(unsigned long long i=g_graph_nodes[tid].starting; i<(g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting); i++)
			{
#if defined (CUDA_HYB)
			unsigned long long id;
              if (i < edge_dev_size)
                id = g_graph_edges[i];
              else
                id = g_graph_edges_2[i - edge_dev_size];
#else
			unsigned long long id = g_graph_edges[i];
#endif
			if(!g_graph_visited[id])
				{
				g_cost[id]=g_cost[tid]+1;
				g_updating_graph_mask[id]=true;
				}
			}
	}
}

#endif 
